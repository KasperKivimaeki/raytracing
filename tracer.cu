#include "tracer.hpp"
#include "rays.hpp"
#include <SDL.h>
#include <hip/hip_runtime.h>

Tracer::Tracer(App *app) {
    buffer = new int[SCREEN_WIDTH * SCREEN_HEIGHT];


    vertices = std::vector<vec3f>();
    triangles = std::vector<vec3i>();
    normals = std::vector<vec3f>();

    this->frame = 0;

    this->app = app;

    // TODO: Fix to support window resizing
    this->fovy = PI/2;
    this->fovx = PI/2;

    loadObjFile(triangles, vertices, normals);

    this->ang = 45;

    this->origin = {0, -200, -200};
    this->click = 0;

    vBuf = NULL;
    nBuf = NULL;
    tBuf = NULL;
    gBuf = NULL;

    CHECK(hipMalloc((void**)&vBuf, vertices.size() * sizeof(vec3f)));
    CHECK(hipMalloc((void**)&nBuf, normals.size() * sizeof(vec3f)));
    CHECK(hipMalloc((void**)&tBuf, 3 * triangles.size() * sizeof(int)));
    CHECK(hipMalloc((void**)&gBuf, SCREEN_HEIGHT * SCREEN_WIDTH * sizeof(int)));
    CHECK(hipMemcpy(nBuf, normals.data(), normals.size() * sizeof(vec3f), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(vBuf, vertices.data(), vertices.size() * sizeof(vec3f), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(tBuf, triangles.data(), 3 * triangles.size() * sizeof(int), hipMemcpyHostToDevice));
}

Tracer::~Tracer() {
    delete[] buffer;

    CHECK(hipFree(nBuf));
    CHECK(hipFree(vBuf));
    CHECK(hipFree(tBuf));
    CHECK(hipFree(gBuf));
}

void Tracer::draw() const {
    dim3 blocks(SCREEN_WIDTH/8, SCREEN_HEIGHT/8);
    dim3 threads(8, 8);

    CHECK(hipMemset(gBuf, 0, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(int)));

    drawRay<<<blocks, threads>>>(origin, ang*PI/180, vBuf, nBuf, triangles.size(), tBuf, triangles.size(), fovx, fovy, gBuf);
    hipDeviceSynchronize();

    CHECK(hipMemcpy(buffer, gBuf, SCREEN_WIDTH * SCREEN_HEIGHT * sizeof(int), hipMemcpyDeviceToHost));
}

void Tracer::run() {
    while(!click) {
        CHECK(hipMemcpy(vBuf, vertices.data(), vertices.size() * sizeof(vec3f), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(tBuf, triangles.data(), 3 * triangles.size() * sizeof(int), hipMemcpyHostToDevice));

        auto start = CURRENT_TIME;
        draw();
        auto end = CURRENT_TIME;
        int ms = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count();
        printf("%dms x: %f, y: %f, z: %f, ang: %d\n", ms, origin.x, origin.y, origin.z, ang);

        app->LDS_prepareScene(frame, buffer);
        getInput();
        app->LDS_presentScene();
        frame += 1;
    }
}

void Tracer::getInput() {
    SDL_Event event;

    while (SDL_PollEvent(&event)) {
        switch (event.type) {
            case SDL_QUIT:
                exit(0);
                break;
            case SDL_MOUSEBUTTONDOWN:
                sy1 = event.button.y;
                sx1 = event.button.x;
                break;
            case SDL_MOUSEWHEEL:
                if (event.wheel.y > 0)
                    origin.y += 0.1;
                if (event.wheel.y < 0)
                    origin.y -= 0.1;
                break;
            case SDL_KEYDOWN:
                switch(event.key.keysym.sym) {
                    case SDLK_ESCAPE:
                        exit(0);
                        break;
                    case 'a':
                    case SDLK_LEFT:
                        origin.x -= 1.0;
                        break;
                    case 'd':
                    case SDLK_RIGHT:
                        origin.x += 1.0;
                        break;
                    case 'w':
                    case SDLK_UP:
                        origin.z += 1.0;
                        break;
                    case 's':
                    case SDLK_DOWN:
                        origin.z -= 1.0;
                        break;
                    case 'q':
                        ang += 5;
                        break;
                    case 'e':
                        ang -= 5;
                        break;
                }
                break;
            default:
                break;
        }
    }
}
