#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <fstream>
#include <sstream>
#include <string>
#include <assert.h>

#include "rays.hpp"
#include "sdl.hpp"

#define IFDEBUG(...)

using std::vector;

__device__ mat3f rotation3d(float ang, int l, int m, int n) {
    mat3f mat;
    mat.a0 = l*l*(1 - cos(ang)) + cos(ang);
    mat.b0 = m*l*(1 - cos(ang)) - sin(ang) * n;
    mat.c0 = n*l*(1 - cos(ang)) + sin(ang) * m;
    mat.a1 = l*m*(1 - cos(ang)) + sin(ang) * n;
    mat.b1 = m*m*(1 - cos(ang)) + cos(ang);
    mat.c1 = n*m*(1 - cos(ang)) - sin(ang) * l;
    mat.a2 = l*n*(1 - cos(ang)) - sin(ang) * m;
    mat.b2 = m*n*(1 - cos(ang)) + sin(ang) * l;
    mat.c2 = n*n*(1 - cos(ang)) + cos(ang);
    return mat;
}

__device__ vec3f multvec3f(vec3f v, mat3f m) {
    vec3f r;

    r.x = v.x * m.a0 + v.y * m.b0 + v.z * m.c0;
    r.y = v.x * m.a1 + v.y * m.b1 + v.z * m.c1;
    r.z = v.x * m.a2 + v.y * m.b2 + v.z * m.c2;

    return r;
}

void check(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << ": "
            << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK(x) check(x, #x)

void loadObjFile(vector<vec3i> &triangles, vector<vec3f> &vertices, vector<vec3f> &normals) {
    std::ifstream ifile("obj/teapot.obj");

    std::string nil;

    std::string line;
    while (std::getline(ifile, line)) {
        std::istringstream iss(line);
        switch(line[0]) {
            case 'v':
                vec3f vertex;
                iss >> nil >> vertex.x >> vertex.y >> vertex.z;
                vertices.push_back(vertex);
                break;
            case 'f':
                vec3i triangle;
                iss >> nil >> triangle.x >> triangle.y >> triangle.z;
                triangles.push_back(triangle);
                break;
            case 'n':
                vec3f normal;
                iss >> nil >> normal.x >> normal.y >> normal.z;
                normals.push_back(normal);
                break;
            default:
                break;
        }
    }

    std::cout << "v: " << vertices.size() << std::endl;
    std::cout << "n: " << normals.size() << std::endl;
    std::cout << "t: " << triangles.size()/3 << std::endl;
}

__global__ void drawRay(vec3f camera, float ang, const vec3f *vertices, const vec3f *normals, int vn, const int *triangles, int tr, double fovx, double fovy, int *buffer) {
    int h = blockIdx.y * 8 + threadIdx.y;
    int w = blockIdx.x * 8 + threadIdx.x;

    // POINT IN IMAGE PLANE
    vec3f p;
    p.x = w*2.0f/SCREEN_WIDTH - 1.f;
    p.y = 1.0f/(tan(fovx/2));
    p.z = h*2.0f/SCREEN_HEIGHT - 1.f;

    mat3f camRot = rotation3d(ang, 1, 0, 0); // l=0, m=0, n=1

    IFDEBUG(abs(p.x) < 1.0f || h || w);
    IFDEBUG(abs(p.z) < 1.0f || h || w);

    // A RAY
    vec3f ray = multvec3f(p, camRot);

    vec3f e1, e2;
    vec3f tvec, pvec, qvec;

    float mt = 0;

    for(int it = 0; it < tr; it++) {
        assert(3*it + 2 < tr*3);

        int t0 = triangles[3*it + 0];
        int t1 = triangles[3*it + 1];
        int t2 = triangles[3*it + 2];

        IFDEBUG(t0 < vn);
        IFDEBUG(t1 < vn);
        IFDEBUG(t2 < vn);

        // Copy three vertices of a triangle
        vec3f v0 = vertices[t0];
        vec3f v1 = vertices[t1];
        vec3f v2 = vertices[t2];

        int intersect = 0;
        double u, v, t, det, inv_det;

        sub(e1, v1, v0);
        sub(e2, v2, v0);

        cross(pvec, ray, e2);

        det = dot(pvec, e1);

        if (det > -EPSILON && det < EPSILON) {
            intersect = 0;
        } else {
            inv_det = 1.0 / det;
            sub(tvec, camera, v0);
            u = dot(tvec, pvec) * inv_det;
            if(u < 0.0 || u > 1.0) {
                intersect = 0;
            } else {
                cross(qvec, tvec, e1);
                v = dot(ray, qvec) * inv_det;
                if(v < 0.0 || u + v > 1.0)
                    intersect = 0;
                else {
                    t = dot(e2, qvec) * inv_det;
                    intersect = t > 0; 
                }
            }
        }

        if (intersect && t > mt) {
            vec3f vn = normals[t0];

            float mul = dot(vn, ray) / 3.0f;
            int col = (int)(255.0 * mul) & 0xff;

            mt = t;

            buffer[w + h*SCREEN_WIDTH] = col | (col << 8) | (col << 16); 
        }
    }
}
